// compile with nvcc hello_world_gpu.cu -o hello_world_gpu
// execute using ./hello_world_gpu


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda_multi(float f) {
    printf("Hello block %d thread %d, f=%f\n", blockIdx.x, threadIdx.x, f);
}

int main() {
    // 5 Blocks with 3 Thread
    // try and execute multiple times and have a look at the order
    hello_cuda_multi<<<5, 3>>>(1.2345f);
    hipDeviceSynchronize();
    return 0;
}