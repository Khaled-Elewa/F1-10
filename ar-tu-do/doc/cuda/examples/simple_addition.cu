// compile with nvcc simple_addition.cu -o simple_addition
// execute using ./simple_addition


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
  *c = *a + *b;
  printf("d_a: %d, d_b: %d, d_c: %d\n", *a, *b, *c);
}

int main(void) {
  int a, b, c;            // host copies of a, b, c
  int *d_a, *d_b, *d_c;   // device copies of a, b, c
  int size = sizeof(int);

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Setup input values
  a = 2;
  b = 7;

  printf("Initial state\n");
  printf("a: %d, b: %d, c: %d\n", a, b, c);
  
  // Copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  printf("Copy to device\n");
  printf("a: %d, b: %d, c: %d\n", a, b, c);
  
  // Launch add() kernel on GPU
  add<<<1,1>>>(d_a, d_b, d_c);

  printf("After kernel launch\n");
  printf("a: %d, b: %d, c: %d\n", a, b, c);
  
  // Copy result back to host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

  printf("After copy to host\n");
  printf("a: %d, b: %d, c: %d\n", a, b, c);
  
  // Cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  printf("After cleanup\n");
  printf("a: %d, b: %d, c: %d\n", a, b, c);
  
  return 0;
}