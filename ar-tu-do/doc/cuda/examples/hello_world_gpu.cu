// compile with nvcc hello_world_gpu.cu -o hello_world_gpu
// execute using ./hello_world_gpu


#include <hip/hip_runtime.h>
#include <stdio.h>

// device code, called kernel
__global__ void mykernel() {
  printf("Hello World from GPU!\n");
}

int main(void) {
  // kernel launch, async
  // 1 Block with 1 Thread
  mykernel<<<1,1>>>();

  // wait for device to finish execution
  hipDeviceSynchronize();

  return 0;
}